
#include <hip/hip_runtime.h>
#include <stdio.h>

inline __host__ __device__ float dot(float3 a, float3 b)
{
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

inline __host__ __device__ float length(float3 v)
{
    return sqrtf(dot(v, v));
}

const float G = 6.6742867e-5f;

__global__ void simGlobalStep(float3* pos, float3* vel, int total) {
  int idx = threadIdx.x + blockIdx.x*blockDim.x;

  if (idx < total) {
    float3 force =  make_float3(0.0f, 0.0f, 0.0f);

    for (int j = 0; j < total; ++j) {
      float3 dlt;
      float sqlen;
      float len;

      if (j == idx) {
        continue;
      }

      dlt.x = pos[j].x - pos[idx].x;
      dlt.y = pos[j].y - pos[idx].y;
      dlt.z = pos[j].z - pos[idx].z;

      sqlen = dot(dlt, dlt);

      len = sqrtf(sqlen);

      dlt.x /= len;
      dlt.y /= len;
      dlt.z /= len;

      sqlen = (sqlen < 1.0f) ? 1.0f : sqlen;

      force.x += dlt.x * G / sqlen;
      force.y += dlt.y * G / sqlen;
      force.z += dlt.z * G / sqlen;
    }

    vel[idx].x += force.x;
    vel[idx].y += force.y;
    vel[idx].z += force.z;
  }
}

void* cudaPosData;
void* cudaVelData;

int simInitialize(int totalSize, void* ipos, void* ivel) {

  hipError_t error;

  error = hipGetLastError();
  if (error != hipSuccess) {
    printf("%s:%d: CUDA: %s (%d)\n", __FILE__, __LINE__, hipGetErrorString(error), error);
    return -1;
  }

  hipMalloc(&cudaPosData, totalSize*sizeof(float3));
  hipDeviceSynchronize();
  error = hipGetLastError();
  if (error != hipSuccess) {
    printf("%s:%d: CUDA: %s (%d)\n", __FILE__, __LINE__, hipGetErrorString(error), error);
    return -1;
  }
  hipMemcpy(cudaPosData, ipos, totalSize*sizeof(float3), hipMemcpyHostToDevice);

  hipMalloc(&cudaVelData, totalSize*sizeof(float3));
  hipDeviceSynchronize();
  error = hipGetLastError();
  if (error != hipSuccess) {
    printf("%s:%d: CUDA: %s (%d)\n", __FILE__, __LINE__, hipGetErrorString(error), error);
    return -1;
  }
  hipMemcpy(cudaVelData, ivel, totalSize*sizeof(float3), hipMemcpyHostToDevice);
  return 0;
}

int simStep(void* inPos, void* outVel, int totalSize) {
  hipError_t error;

  hipMemcpy(cudaPosData, inPos, totalSize*sizeof(float3), hipMemcpyHostToDevice);
  simGlobalStep<<<totalSize/512,512>>>((float3*)cudaPosData, (float3*)cudaVelData, totalSize);
  hipMemcpy(outVel, cudaVelData, totalSize*sizeof(float3), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  error = hipGetLastError();
  if (error != hipSuccess) {
    printf("%s:%d: CUDA: %s (%d)\n", __FILE__, __LINE__, hipGetErrorString(error), error);
    return -1;
  }

  return 0;
}

int simCleanup() {
  hipFree(cudaPosData);
  hipFree(cudaVelData);
  return 0;
}